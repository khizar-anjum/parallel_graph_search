#include "hip/hip_runtime.h"
#include "dijkstra.cuh"
//#include "pqueue.cu"

int* dijkstra_par(graph &g, int src){
	// starts at the source and calculates the distance for all the 
	// vertices inside the graph

	// check if src exists and populate it
	std::map<int, int>::iterator it = g.name_to_vertex.find(src);
	if(it == g.name_to_vertex.end()) throw std::invalid_argument( "Invalid vertex name" );
	int src_vertex = it->second;

	// initialize everything
	bool finished = false;
	int NUM_QUEUES = 1;
	bool* visited = new bool[g.num_vertices];
	int* costs = new int[g.num_vertices];
	int* size_h = new int[NUM_QUEUES];
	unsigned int* SWsize; 
	unsigned int* Tsize;
	unsigned int* SWsize_h = new unsigned int[1]; SWsize_h[0] = 0; 
	unsigned int* Tsize_h = new unsigned int[1]; Tsize_h[0] = 0;
	for(int i = 0; i < g.num_vertices; i++){
		visited[i] = false; //visited flags for vertices
		costs[i] = 0x70000000; //costs for each vertex
	}
	costs[src_vertex] = 0;
	for(int i = 0; i < NUM_QUEUES; i++){
		size_h[i] = 0;
	}


	int* weight_arr_d; int* connected_to_d; int* costs_d; 
	size_t bytes = g.num_edges * sizeof(int);
	hipMalloc(&weight_arr_d, bytes);
	hipMalloc(&connected_to_d, bytes);

	int* heap; int* size; int* S; int* W; int* T; 
	bool* visited_d; int* num_connected_d; int* index_arr_d;
	size_t vbytes = g.num_vertices * sizeof(int);
	hipMalloc(&heap, 2*vbytes*NUM_QUEUES);
	hipMalloc(&size, NUM_QUEUES*sizeof(int));
	hipMalloc(&num_connected_d, vbytes);
	hipMalloc(&index_arr_d, vbytes);
	hipMalloc(&costs_d, vbytes);
	hipMalloc(&S, vbytes);
	hipMalloc(&W, vbytes);
	hipMalloc(&T, vbytes);
	hipMalloc(&SWsize, sizeof(unsigned int));
	hipMalloc(&Tsize, sizeof(unsigned int));
	hipMalloc(&visited_d, g.num_vertices*sizeof(bool));

	//lets copy all the stuff from device memory to host memory
	hipMemcpy(size, size_h, NUM_QUEUES*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(weight_arr_d, g.weight_arr, bytes, hipMemcpyHostToDevice);
	hipMemcpy(index_arr_d, g.index_arr, vbytes, hipMemcpyHostToDevice);
	hipMemcpy(connected_to_d, g.connected_to, bytes, hipMemcpyHostToDevice);
	hipMemcpy(costs_d, costs, vbytes, hipMemcpyHostToDevice);
	hipMemcpy(visited_d, visited, g.num_vertices*sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(num_connected_d, g.num_connected, vbytes, hipMemcpyHostToDevice);
	insertKernelDijkstra <<<1,1>>> (heap, size, src_vertex, 0, g.num_vertices);
	
	//Now, extracting mins and carrying on till we find the destination!
	while(!finished){
		//Extracting min and pulling the corresponding vertex into cloud
		hipMemcpy(SWsize, SWsize_h, sizeof(unsigned int), hipMemcpyHostToDevice);
		extractKernelDijkstra<<<1, NUM_QUEUES>>>(heap, size, S, W, SWsize, num_connected_d, weight_arr_d,
			costs_d, index_arr_d, connected_to_d, NUM_QUEUES, g.num_vertices);
		hipDeviceSynchronize();

		hipMemcpy(Tsize, Tsize_h, sizeof(unsigned int), hipMemcpyHostToDevice);
		deduplicateKernelDijkstra<<<1, NUM_QUEUES>>>(S, W, SWsize, T, Tsize, visited_d, costs_d, g.num_vertices);
		hipDeviceSynchronize();

		computeKernelDijkstra<<<1, NUM_QUEUES>>>(heap, size, T, Tsize, visited_d, costs_d, g.num_vertices);
		hipDeviceSynchronize();

		//Exit when all the vertices explored 
		//meaning that all queues are empty
		hipMemcpy(size_h, size, NUM_QUEUES*sizeof(int), hipMemcpyDeviceToHost);
		for(int i = 0; i < NUM_QUEUES; i++){
			//printf("%d\n", size_h[i]);
			if(size_h[i] > 1)
				break;	
			finished = true;
		}
	}

	hipMemcpy(costs, costs_d, vbytes, hipMemcpyDeviceToHost);
	hipFree(heap); hipFree(size); hipFree(num_connected_d); hipFree(index_arr_d);
	hipFree(costs_d); hipFree(S); hipFree(W); hipFree(T); hipFree(visited_d);
	hipFree(weight_arr_d); hipFree(connected_to_d); hipFree(SWsize); hipFree(Tsize);
	delete [] visited;

	return costs;
}

__global__ void insertKernelDijkstra(int* heap, int* size, int src_vertex, int priority, int num_vertices){
	//kernel to insert elements
	insert_GPU(heap, size[0], src_vertex, priority, num_vertices);
}

__global__ void extractKernelDijkstra(int* heap, int* size, int* S, int* W, unsigned int* SWsize, int* num_connected_d, int* weight_arr_d,
								int* costs_d, int* index_arr_d, int* connected_to_d, int NUM_QUEUES, int num_vertices){
	//kernel to extract elements from queues. Here num_threads must be equal to NUM_QUEUES
	int current_vertex;
	int current_weight;
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int oldSWsize = 0;
	if(size[index] > 0){
		ExtractMin_GPU(&heap[2*index*num_vertices], size[index], current_vertex, current_weight, num_vertices);
		//printf("%d %d\n", current_vertex, current_weight);
		for(int i = 0; i < num_connected_d[current_vertex]; i++){
			oldSWsize = atomicAdd(SWsize, 1);
			atomicExch(&S[oldSWsize], connected_to_d[index_arr_d[current_vertex] + i]);
			W[oldSWsize] = weight_arr_d[index_arr_d[current_vertex] + i] + costs_d[current_vertex];
		}
	}
}

__global__ void deduplicateKernelDijkstra(int* S, int* W, unsigned int* SWsize, int* T, unsigned int* Tsize, bool* visited_d, int* costs_d, int num_vertices){
	//kernel to deduplicate elements in S. 
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;
	unsigned int oldTsize;
	
	for(int i = index; i < SWsize[0]; i+=stride){
		//printf("%d %d %d %d %d\n", i, S[i], visited_d[S[i]], costs_d[S[i]], W[i]);
		if((visited_d[S[i]] == true) && (costs_d[S[i]] <= W[i])){
			continue;
		}
		else{
			oldTsize = atomicAdd(Tsize, 1);
			T[oldTsize] = S[i];
			if(costs_d[S[i]] > W[i]){
				atomicExch(&costs_d[S[i]], W[i]);
			}
		}
	}
}

__global__ void computeKernelDijkstra(int* heap, int* size, int* T, unsigned int* Tsize, bool* visited_d, int* costs_d, int num_vertices){
	//kernel to computee elements in T. Since this accesses queues, num_threads must be equal to NUM_QUEUES
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	//printf("%d\n", Tsize[0]);
	for(int i = index; i < Tsize[0]; i+= stride){
		insert_GPU(&heap[2*index*num_vertices], size[index], T[i], costs_d[T[i]], num_vertices);
		visited_d[T[i]] = true;
	}
}
